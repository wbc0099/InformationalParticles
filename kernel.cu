#include "hip/hip_runtime.h"
﻿#include <hipfft/hipfftXt.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <stdio.h>
#include <vector>
#include <string>
#include <time.h>
#include <math.h>
#include <random> 
#include <iomanip>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

//if no run in a long time ,maybe beacause box is too small

//Definitions=======================================================================
// Define the precision of real numbers, could be real/double.
#define real double
#define Pi 3.1415926535897932384626433832795
#define Zero 0
using namespace std;

struct Particle {
    real* x;//save x position in GPU
    real* y;//save y position in GPU
    int* cellX;//save xth cell of nth particle
    int* cellY;//save yth cell of nth particle
    int* cellList;//cell particle id for all particle, as [maxParticlePerCell*id + offsetsCL]?????
    int* cellOffsetsCL;//offset of every cell list to save particle number in this cell 
    int* particleAroundId;//save ids around this on particle, use rd to judge wether is "around"????????
    int* particleAroundFlagX;//mask whether cell of idth particle at the edge of box
    int* particleAroundFlagY;//mask whether cell of idth particle at the edge of box
    int* offsetsAL;//offset of every particle's around list
    int* offsetsNL;//offset of every particle's neighbor list to save neighbor particle id
    int* NeighborList;//neighbor list
    int* NeighborListFlagX;//translate from particleAroundFlagX
    int* NeighborListFlagY;//translate from particleAroundFlagY
    real* fx;//force on the x direction
    real* fy;//force on the y direction
    real* x0ToUpdateHybridList;//save xGpu[id] to judge whether update hybrid list 
    real* y0ToUpdateHybridList;//save yGpu[id] to judge whether update hybrid list
    hiprandState* state;//save random number generator
    int* aroundNum;
    real* kBT;
} PT, pt;//pt is saved in CPU and PT in GPU

struct Parameter {
    real boxX;//box size X
    real boxY;//box size Y
    real cellSizeX;//cell size in the x direction
    real cellSizeY;//cell size in the y direction
    int cellNumX;//num of cell in the x direction
    int cellNumY;//num of cell in the y direction
    int maxParticlePerCell;//theory maxmum particle number in one cell
    real rd;//deadline distance to get NeighborList
    real miniInstanceBetweenParticle;//theory minimum distance from two particle
    real r0;//balance position
    real epsilon;//coefficient of force
    float kBT;//kB*T
    real gammaValue;//Viscosity coefficien
    real rOutUpdateList;//update hybrid list when any one particle move a distance greater than rOutUpdateList
    int particleNum; //particle number 
    real tStart;//start time
    real tStop;//stop time
    real tStep;//step time
    real tExpo;//export data every tExpo
    unsigned long long seed;//generate random number seed
    int blockNum;//num of block in one grid
    int threadNum;//num of thread in one block
    int nthGPU;//which GPU to use
    real rOff;//offset distance to judge whether particle is "around"
    real rOffIn;//Inner diameter of circular ring
    real N;//coefficient of temperture change
    real forceCoefficient;//coefficient of force
    real rNeighborList;//size neighbor
    int kBTChangeMode;
    int kBTChangePM0;
    real visionConeXLen;
} PM;

__device__ int updateListFlag = 0;
__device__ int wrongFlag = 0;
int updateListFlagHost = 0;
int wrongFlagHost = 0;

void ExpoConf(const std::string& str_t);
void MemFree();
void getInput();
void MemAlloc();
void printInput();
void Init_Coords(int flag, Particle pt, Parameter PM);
void initAroundNum(Particle PT, Parameter PM);
void InitOffset();
void HostUpdataToDevice();
void DeviceUpdataToHost();
void listUpdate(Particle PT,Parameter PM);
void forceAndPositionUpdate(Particle PT, Parameter PM);
void iterate(Particle PT,Parameter PM);
void initBlockAndThreadNum();
void showProgress(real tNow, real tStart, real tStop, clock_t clockNow, clock_t clockStart);
int setDevice(int n);
int printGpuError();
int initAll();
__global__ void initState(hiprandState* state,unsigned long long seed, int particleNum);
__global__ void getCellList(Particle PT, Parameter PM);
__global__ void getAroundCellParticleId(Particle PT, Parameter PM);
__global__ void saveXY0ToUpdateHybridList(Particle PT, Parameter PM);
__global__ void checkUpdate(Particle PT, Parameter PM);
__global__ void getForce (Particle PT, Parameter PM);
__global__ void updatePosition(Particle PT, Parameter PM);
__device__ int getNeighborListTry(real x0, real y0, real x1, real y1, Parameter PM);
__device__ int sign(real x);
__device__ int sign01(real x);
__device__ real force (real forceCoefficient,real dr,real rd);
__device__ real generateNormal(hiprandState* state);
__device__ void updateKBT(Particle PT, Parameter PM, int id);

int main()
{
    real tNow = PM.tStart;
    if(!initAll())return 0;    
    ExpoConf("0");

    listUpdate(PT, PM);
    hipDeviceSynchronize();

    if(!printGpuError())return 0; 

    clock_t clockStart = clock();
    for (tNow = PM.tStart;tNow < PM.tStop;tNow += PM.tStep) {
        iterate(PT, PM);

        hipMemcpyFromSymbol(&wrongFlagHost, HIP_SYMBOL(wrongFlag), sizeof(int));
        if (wrongFlagHost == 1)return 0;

        if (floor(tNow / PM.tExpo) > floor((tNow - PM.tStep) / PM.tExpo)) {
            showProgress(tNow, PM.tStart, PM.tStop, clock(), clockStart);
            DeviceUpdataToHost();//下载数据到主机
            int te = floor(tNow / PM.tExpo) + 1;
            string str_t = to_string(te);
            ExpoConf(str_t);
        }
    }

    if(!printGpuError())return 0; 

    MemFree();//释放内存
    hipDeviceReset();
    return 0; // 返回成功状态
}

int initAll(){
    getInput();
    if(!setDevice(PM.nthGPU))return 0;
    MemAlloc();
    printInput();
    Init_Coords(1, pt, PM);
    initBlockAndThreadNum();
    InitOffset();
    initState << <PM.blockNum, PM.threadNum >> > (PT.state, PM.seed, PM.particleNum);
    hipDeviceSynchronize();
    HostUpdataToDevice();
    PM.seed = static_cast<unsigned long long>(time(0));
    printf("seed:%d\n", PM.seed);
    return 1;
}

void getInput() {
    std::ifstream InputFile("input.dat");

    if (!InputFile.is_open()) {
        std::cerr << "Error opening input file!" << std::endl;
        return; // 退出函数
    }

    std::string line;
    int lineCount = 0;

    while (std::getline(InputFile, line)) {
        // 检查是否为注释行
        if (line.empty() || line.find('#') != std::string::npos) {
            continue; // 跳过空行和注释行
        }

        std::istringstream iss(line);
        switch (lineCount) {
        case 0: iss >> PM.boxX; break;
        case 1: iss >> PM.boxY; break;
        case 2: iss >> PM.cellSizeX; break;
        case 3: iss >> PM.cellSizeY; break;
        case 4: iss >> PM.cellNumX; break;
        case 5: iss >> PM.cellNumY; break;
        case 6: iss >> PM.maxParticlePerCell; break;
        case 7: iss >> PM.rd; break;
        case 8: iss >> PM.miniInstanceBetweenParticle; break;
        case 9: iss >> PM.r0; break;
        case 10: iss >> PM.epsilon; break;
        case 11: iss >> PM.kBT; break;
        case 12: iss >> PM.gammaValue; break;
        case 13: iss >> PM.rOutUpdateList; break;
        case 14: iss >> PM.particleNum; break;
        case 15: iss >> PM.tStart; break;
        case 16: iss >> PM.tStop; break;
        case 17: iss >> PM.tStep; break;
        case 18: iss >> PM.tExpo; break;
	    case 19: iss >> PM.nthGPU; break;
        case 20: iss >> PM.rOff; break;
        case 21: iss >> PM.rOffIn; break;
        case 22: iss >> PM.N; break;
        case 23: iss >> PM.forceCoefficient; break;
        case 24: iss >> PM.rNeighborList; break;
        case 25: iss >> PM.kBTChangeMode; break;
        case 26: iss >> PM.kBTChangePM0; break;
        case 27: iss >> PM.visionConeXLen; break;
        default: break; // 超过预期行数时不处理
        }
        lineCount++;
    }

    InputFile.close();
}

int setDevice(int n){
    hipError_t err = hipSetDevice(n);
    if (err != hipSuccess) {
        std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
        return 0;
    }
    return 1;
}

void printInput() {
    std::cout << "Box X: " << PM.boxX << std::endl;
    std::cout << "Box Y: " << PM.boxY << std::endl;
    std::cout << "Cell size X: " << PM.cellSizeX << std::endl;
    std::cout << "Cell size Y: " << PM.cellSizeY << std::endl;
    std::cout << "Cell num X: " << PM.cellNumX << std::endl;
    std::cout << "Cell num Y: " << PM.cellNumY << std::endl;
    std::cout << "Max particle per cell: " << PM.maxParticlePerCell << std::endl;
    std::cout << "Deadline distance: " << PM.rd << std::endl;
    std::cout << "Mini instance between particle: " << PM.miniInstanceBetweenParticle << std::endl;
    std::cout << "Equilibrium position: " << PM.r0 << std::endl;
    std::cout << "Epsilon: " << PM.epsilon << std::endl;
    std::cout << "kBT: " << PM.kBT << std::endl;
    std::cout << "Gamma value: " << PM.gammaValue << std::endl;
    std::cout << "Update list distance threshold: " << PM.rOutUpdateList << std::endl;
    std::cout << "Particle num: " << PM.particleNum << std::endl;
    std::cout << "Start time: " << PM.tStart << std::endl;
    std::cout << "Stop time: " << PM.tStop << std::endl;
    std::cout << "Time step: " << PM.tStep << std::endl;
    std::cout << "TExpo: " << PM.tExpo << std::endl;
    std::cout << "nthGPU: " << PM.nthGPU << std::endl;
    std::cout << "rOff: " << PM.rOff << std::endl;
    std::cout << "rOffInner: " << PM.rOffIn << std::endl;
    std::cout << "N: " << PM.N << std::endl;
    std::cout << "forceCoefficient: " << PM.forceCoefficient<< std::endl;
    std::cout << "rNeighborList: " << PM.rNeighborList<< std::endl;
    std::cout << "kBTChangeMode: " << PM.kBTChangeMode<< std::endl;
    std::cout << "kBTChangePM0: " << PM.kBTChangePM0<< std::endl;
    std::cout << "visionConeXLen: " << PM.visionConeXLen<< std::endl;
}

void Init_Coords(int flag, Particle pt, Parameter PM) {
    memset(pt.cellList, 0, PM.cellNumX * PM.cellNumY * PM.maxParticlePerCell * sizeof(int));
    memset(pt.cellOffsetsCL, 0, PM.cellNumX * PM.cellNumY * PM.maxParticlePerCell * sizeof(int));
    int N=PM.particleNum;
    real xBox = PM.boxX;
    real yBox = PM.boxY;
    std::default_random_engine e;
    std::uniform_real_distribution<double> u(0.0, 1.0);
    e.seed(time(0));
    real x0, y0, dx, dy;
    int cellX,cellY,cellX1,cellY1,cellAround;
    int xFlag,yFlag;
    int wrongFlag=0;
    for(int i=0;i<N;i++){
        while(1){
            x0=u(e)*xBox;
            y0=u(e)*yBox;
            //printf("id:%d,x:%f,y:%f\n",i,x0,y0);
            wrongFlag=0;
            cellX=std::floor(x0/PM.cellSizeX);
            cellY=std::floor(y0/PM.cellSizeY);
            for(int x=-1;x<=1;x++){
                for(int y=-1;y<=1;y++){
                    if(cellX+x==-1){
                        cellX1=PM.cellNumX-1;
                        xFlag=1;
                    }else if(cellX+x==PM.cellNumX){
                        cellX1=0;
                        xFlag=-1;
                    }else{
                        cellX1=cellX+x;
                        xFlag=0;
                    }
                    if(cellY+y==-1){
                        cellY1=PM.cellNumY-1;
                        yFlag=1;
                    }else if(cellY+y==PM.cellNumY){                    
                        cellY1=0;
                        yFlag=-1;
                    }else{
                        cellY1=cellY+y;
                        yFlag=0;
                    }
                    cellAround=cellX1+cellY1*PM.cellNumX;
                    for(int j=0;j<pt.cellOffsetsCL[cellAround];j++){
                        //printf("cell:%d,cellAround:%d,j:%d,x0:%f,y0:%f,x:%f,y:%f\n",cellX+cellY*PM.cellNumX,cellAround,j,x0,y0,pt.x[pt.cellList[cellAround*PM.maxParticlePerCell+j]],pt.y[pt.cellList[cellAround*PM.maxParticlePerCell+j]]);
                        dx=(x0-pt.x[pt.cellList[cellAround*PM.maxParticlePerCell+j]])+xFlag*PM.boxX;
                        dy=(y0-pt.y[pt.cellList[cellAround*PM.maxParticlePerCell+j]])+yFlag*PM.boxY;
                        if(dx*dx+dy*dy<PM.r0*PM.r0){
                            wrongFlag=1;
                            break;
                        }
                    }
                    if(wrongFlag==1){
                        break;
                    }
                }
                if(wrongFlag==1){ 
                    break;
                }
            }
            if(wrongFlag==0){
                break;
            }else continue;
        }        
        pt.x[i]=x0;
        pt.y[i]=y0;
        pt.cellList[(cellX+cellY*PM.cellNumX)*PM.maxParticlePerCell+pt.cellOffsetsCL[cellX+cellY*PM.cellNumX]]=i;
        pt.cellOffsetsCL[cellX+cellY*PM.cellNumX]++;
    }
}

__global__ void initState(hiprandState* state,unsigned long long seed, int particleNum) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= particleNum)return;
    hiprand_init(seed, id, 0, &state[id]);
}

void initBlockAndThreadNum() {
    PM.threadNum = 256;
    PM.blockNum = (PM.particleNum + PM.threadNum - 1) / PM.threadNum;
    printf("blockNum:%d,threadNum:%d\n", PM.blockNum, PM.threadNum);
}

void InitOffset() {
    hipMemset(PT.cellOffsetsCL, 0, sizeof(int) * PM.cellNumX * PM.cellNumY * PM.maxParticlePerCell);
    hipMemset(PT.offsetsNL, 0, sizeof(int) * PM.particleNum);
    hipMemset(PT.offsetsAL, 0, sizeof(int) * PM.particleNum);
}

void HostUpdataToDevice() {
    hipMemcpy(PT.x, pt.x, PM.particleNum * sizeof(real), hipMemcpyHostToDevice);
    hipMemcpy(PT.y, pt.y, PM.particleNum * sizeof(real), hipMemcpyHostToDevice);
}

void DeviceUpdataToHost() {
    hipMemcpy(pt.x, PT.x, PM.particleNum * sizeof(real), hipMemcpyDeviceToHost);
    hipMemcpy(pt.y, PT.y, PM.particleNum * sizeof(real), hipMemcpyDeviceToHost);
    hipMemcpy(pt.kBT, PT.kBT, PM.particleNum * sizeof(real), hipMemcpyDeviceToHost);
    hipMemcpy(pt.aroundNum, PT.aroundNum, PM.particleNum * sizeof(int), hipMemcpyDeviceToHost);
}

void MemAlloc() {
    // Allocate particle mem in host memory.
    pt.x = new real[PM.particleNum];
    pt.y = new real[PM.particleNum];
    pt.cellList = new int[PM.cellNumX * PM.cellNumY * PM.maxParticlePerCell];
    pt.cellOffsetsCL = new int[PM.cellNumX * PM.cellNumY * PM.maxParticlePerCell];
    pt.particleAroundId = new int[9 * PM.particleNum * PM.maxParticlePerCell];
    pt.particleAroundFlagX = new int[9 * PM.particleNum * PM.maxParticlePerCell];
    pt.particleAroundFlagY = new int[9 * PM.particleNum * PM.maxParticlePerCell];
    pt.offsetsNL = new int[PM.particleNum];
    pt.NeighborList = new int[PM.particleNum * PM.maxParticlePerCell];
    pt.NeighborListFlagX = new int[PM.particleNum];
    pt.NeighborListFlagY = new int[PM.particleNum];
    pt.fx = new real[PM.particleNum];
    pt.fy = new real[PM.particleNum];
    pt.x0ToUpdateHybridList = new real[PM.particleNum];
    pt.y0ToUpdateHybridList = new real[PM.particleNum];
    pt.state = new hiprandState[PM.particleNum];
    pt.aroundNum = new int[PM.particleNum];
    pt.kBT = new real[PM.particleNum];


    // Allocate memory of fields in device.
    hipMalloc((void**)&PT.x, PM.particleNum * sizeof(real));
    hipMalloc((void**)&PT.y, PM.particleNum * sizeof(real));
    hipMalloc((void**)&PT.cellX, PM.particleNum * sizeof(int));
    hipMalloc((void**)&PT.cellY, PM.particleNum * sizeof(int));
    hipMalloc((void**)&PT.cellList, PM.cellNumX * PM.cellNumY * PM.maxParticlePerCell * sizeof(int));
    hipMalloc((void**)&PT.cellOffsetsCL, PM.cellNumX * PM.cellNumY * PM.maxParticlePerCell * sizeof(int));
    hipMalloc((void**)&PT.particleAroundId, 9 * PM.maxParticlePerCell * PM.particleNum * sizeof(int));
    hipMalloc((void**)&PT.particleAroundFlagX, 9 * PM.particleNum * sizeof(int));
    hipMalloc((void**)&PT.particleAroundFlagY, 9 * PM.particleNum * sizeof(int));
    hipMalloc((void**)&PT.offsetsAL, PM.particleNum * sizeof(int));
    hipMalloc((void**)&PT.offsetsNL, PM.particleNum * sizeof(int));
    hipMalloc((void**)&PT.NeighborList, PM.particleNum * PM.maxParticlePerCell * sizeof(int));
    hipMalloc((void**)&PT.NeighborListFlagX, PM.particleNum * PM.maxParticlePerCell * sizeof(int));
    hipMalloc((void**)&PT.NeighborListFlagY, PM.maxParticlePerCell * PM.particleNum * sizeof(int));
    hipMalloc((void**)&PT.fx, PM.particleNum * sizeof(real));
    hipMalloc((void**)&PT.fy, PM.particleNum * sizeof(real));
    hipMalloc((void**)&PT.x0ToUpdateHybridList, PM.particleNum * sizeof(real));
    hipMalloc((void**)&PT.y0ToUpdateHybridList, PM.particleNum * sizeof(real));
    hipMalloc((void**)&PT.state, PM.particleNum * sizeof(hiprandState));
    hipMalloc((void**)&PT.aroundNum, PM.particleNum * sizeof(int));
    hipMalloc((void**)&PT.kBT, PM.particleNum * sizeof(real));

    hipMemset(PT.aroundNum, PM.kBT, PM.particleNum * sizeof(int));
    
}

void ExpoConf(const std::string& str_t) {
    std::ofstream ConfFile;
    //设置输出精度
    int PrecData = 8;

    // 文件名
    std::string ConfFileName = "conf_" + str_t + ".dat";
    ConfFile.open(ConfFileName.c_str());

    if (!ConfFile.is_open()) {
        std::cerr << "无法打开文件: " << ConfFileName << std::endl;
        return;
    }
    for (int idx = 0; idx < PM.particleNum; idx++) {
        // 使用固定格式和精度输出数据
        ConfFile << std::fixed << std::setprecision(PrecData)
            << pt.x[idx] << ' '
            << pt.y[idx] << ' '
            << pt.kBT[idx] << ' '
            << pt.aroundNum[idx];
        ConfFile << std::endl; // 换行
    }
    ConfFile.close();
}

void listUpdate(Particle PT,Parameter PM) {
    InitOffset();
    getCellList << <PM.blockNum, PM.threadNum >> > (PT, PM);
    hipDeviceSynchronize();
    getAroundCellParticleId << <PM.blockNum, PM.threadNum >> > (PT, PM);
    hipDeviceSynchronize();
    saveXY0ToUpdateHybridList << <PM.blockNum, PM.threadNum >> > (PT, PM);
    hipDeviceSynchronize();
}

__global__ void getCellList(Particle PT, Parameter PM) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= PM.particleNum)return;
    PT.cellX[id] = std::floor(PT.x[id] / PM.cellSizeX);
    PT.cellY[id] = std::floor(PT.y[id] / PM.cellSizeY);
    int cellId = PT.cellY[id] * PM.cellNumX + PT.cellX[id];
    int offsetsCL = atomicAdd(&PT.cellOffsetsCL[cellId], 1);
    if (offsetsCL < PM.maxParticlePerCell) {
        PT.cellList[cellId * PM.maxParticlePerCell + offsetsCL] = id;
    }
    else {
        printf("wrong: offsetsCL is greater than maxParticlePerCell");//append cout error later
        wrongFlag=1;
    }
}

__global__ void getAroundCellParticleId(Particle PT, Parameter PM) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= PM.particleNum)return;
    int offsetPAI = 0;//particleAroundId put particleId in PAI
    int periodicBoundaryFlagX, periodicBoundaryFlagY;
    int cellXAround, cellYAround;
    int cellAroundId;
    for (int x = -1;x <= 1;x++) {
        for (int y = -1;y <= 1;y++) {
            if (PT.cellX[id] + x == -1) {
                cellXAround = PM.cellNumX - 1;
                periodicBoundaryFlagX = 1;
            }
            else if (PT.cellX[id] + x == PM.cellNumX) {
                cellXAround = 0;
                periodicBoundaryFlagX = -1;

            }
            else {
                cellXAround = PT.cellX[id] + x;
                periodicBoundaryFlagX = 0;
            }
            if (PT.cellY[id] + y == -1) {
                cellYAround = PM.cellNumY - 1;
                periodicBoundaryFlagY = 1;
            }
            else if (PT.cellY[id] + y == PM.cellNumY) {
                cellYAround = 0;
                periodicBoundaryFlagY = -1;
            }
            else {
                cellYAround = PT.cellY[id] + y;
                periodicBoundaryFlagY = 0;
            }
            int cellAroundId = cellYAround * PM.cellNumX + cellXAround;

            for (int i = 0;i < PT.cellOffsetsCL[cellAroundId];i++) {
                if (PT.cellList[cellAroundId * PM.maxParticlePerCell + i] == id)continue;
                int ifNeighbor = getNeighborListTry(PT.x[id], PT.y[id], PT.x[PT.cellList[cellAroundId * PM.maxParticlePerCell + i]]\
                    , PT.y[PT.cellList[cellAroundId * PM.maxParticlePerCell + i]], PM);
                if (ifNeighbor) {
                    PT.NeighborList[id * PM.maxParticlePerCell + PT.offsetsNL[id]] = PT.cellList[cellAroundId * PM.maxParticlePerCell + i];
                    PT.NeighborListFlagX[id * PM.maxParticlePerCell + PT.offsetsNL[id]] = periodicBoundaryFlagX;
                    PT.NeighborListFlagY[id * PM.maxParticlePerCell + PT.offsetsNL[id]] = periodicBoundaryFlagY;//nodebug
                    atomicAdd(&PT.offsetsNL[id], 1);
                }
            }
        }
    }
}

__device__ int getNeighborListTry(real x0, real y0, real x1, real y1, Parameter PM) {
    real dx = sign(x1 - x0) * (x1 - x0);
    real dy = sign(y1 - y0) * (y1 - y0);
    dx = sign01(0.5 * PM.boxX - dx) * dx + sign01(dx - 0.5 * PM.boxX) * (PM.boxX - dx);
    dy = sign01(0.5 * PM.boxY - dy) * dy + sign01(dy - 0.5 * PM.boxY) * (PM.boxY - dy);
    real dr2 = dx * dx + dy * dy;
    if (dr2 < PM.rNeighborList * PM.rNeighborList) return 1;
    else return 0;
}

__global__ void saveXY0ToUpdateHybridList(Particle PT, Parameter PM) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= PM.particleNum)return;
    PT.x0ToUpdateHybridList[id] = PT.x[id];
    PT.y0ToUpdateHybridList[id] = PT.y[id];
}

int printGpuError(){
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return 0;
    }
    return 1;
}

void iterate(Particle PT,Parameter PM) {
    forceAndPositionUpdate(PT,PM);
    checkUpdate << <PM.blockNum, PM.threadNum >> > (PT, PM);
    hipMemcpyFromSymbol(&updateListFlagHost, HIP_SYMBOL(updateListFlag), sizeof(int));
    if (updateListFlagHost){
        listUpdate(PT, PM);
        updateListFlagHost = 0;
        hipMemcpyToSymbol(HIP_SYMBOL(updateListFlag), &updateListFlagHost, sizeof(int));
    }
}

void forceAndPositionUpdate(Particle PT, Parameter PM) {
    initAroundNum(PT, PM);
    getForce << <PM.blockNum, PM.threadNum >> > (PT, PM);
    hipDeviceSynchronize();
    updatePosition << <PM.blockNum, PM.threadNum >> > (PT, PM);
    hipDeviceSynchronize();
}

void initAroundNum(Particle PT, Parameter PM) {
    hipMemset(PT.aroundNum, 0, sizeof(int) * PM.particleNum);
}

__global__ void getForce (Particle PT, Parameter PM) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= PM.particleNum)return;
    real x0, y0, x1, y1, dx, dy, dr, f12;
    PT.fx[id] = 0;
    PT.fy[id] = 0;
    int i;
    PT.aroundNum[id] = 0;
    for (i = 0;i < PT.offsetsNL[id];i++) {
        x0 = PT.x[id]<PM.boxX-0.5 ? PT.x[id]+0.5:PT.x[id]+0.5-PM.boxX;
        y0 = PT.y[id];
        x1 = PT.x[PT.NeighborList[id * PM.maxParticlePerCell + i]];
        y1 = PT.y[PT.NeighborList[id * PM.maxParticlePerCell + i]];
        dx = sign01(0.5 * PM.boxX - x0 + x1) * sign01(0.5 * PM.boxX + x0 - x1) * (x0 - x1) + \
            sign01(sign(x0 - x1) * (x0 - x1) - 0.5 * PM.boxX) * -sign(x0 - x1) * (PM.boxX - sign(x0 - x1) * (x0 - x1));
        dy = sign01(0.5 * PM.boxY - y0 + y1) * sign01(0.5 * PM.boxY + y0 - y1) * (y0 - y1) + \
            sign01(sign(y0 - y1) * (y0 - y1) - 0.5 * PM.boxY) * -sign(y0 - y1) * (PM.boxY - sign(y0 - y1) * (y0 - y1));
        dr = sqrt(dx * dx + dy * dy);
        
        if(dr<PM.rOff && dx>(dr*PM.visionConeXLen) && dr>PM.rOffIn){
            PT.aroundNum[id] += 1;
        }

       f12=0;
        
        if (PT.fx[id] > 10000 || PT.fx[id] < -10000 || PT.fy[id] > 10000 || PT.fy[id] < -10000) {
            break;
        }
    }
    if (PT.fx[id] > 10000 || PT.fx[id] < -10000) {
        printf("wrong!!!!!!!!!id:%d,fx:%f,fy:%f,dx:%f,dy:%f,x0:%f,x1:%f,y0:%f,y1:%f,NLFX:%d\n", id,PT.fx[id], PT.fy[id], dx,dy,x0,x1, y0, y1, PT.NeighborListFlagX[id * PM.maxParticlePerCell + i]);
        wrongFlag = 1;
    }
    if (PT.fy[id] > 10000 || PT.fy[id] < -10000) {
        printf("wrong!!!!!!!!!id:%d,fx:%f,fy:%f,dx:%f,dy:%f,y0:%f,y1:%f,x0:%f,x1:%f,NLFY:%d\n", id, PT.fx[id], PT.fy[id],dx,dy,y0,y1, x0, x1, PT.NeighborListFlagY[id * PM.maxParticlePerCell + i]);
        wrongFlag = 1;
    }
}

__device__ real force (real forceCoefficient,real dr,real rd){
    return forceCoefficient*(rd-dr)*(rd-dr);
}

__device__ void updateKBT(Particle PT, Parameter PM, int id){
    if (PM.kBTChangeMode == 1){
        PT.kBT[id]=PM.kBT+PM.N*(PT.aroundNum[id]-PM.kBTChangePM0)*(PT.aroundNum[id]-PM.kBTChangePM0)/10;
    } else if (PM.kBTChangeMode == 2){
        PT.kBT[id]=PM.kBT+PM.N*sign(PT.aroundNum[id]-PM.kBTChangePM0)*(PT.aroundNum[id]-PM.kBTChangePM0);
    } else if (PM.kBTChangeMode == 3){
        PT.kBT[id]=PM.kBT+PM.N*(sin((PT.aroundNum[id]-PM.kBTChangePM0)*Pi/3-Pi/2)+1);
    } 
}

__global__ void updatePosition(Particle PT, Parameter PM) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= PM.particleNum)return;
    updateKBT(PT,PM,id);
    real fT = sqrt(2 * PT.kBT[id] * PM.gammaValue * PM.tStep);
    real FRx = generateNormal(&PT.state[id]);
    real FRy = generateNormal(&PT.state[id]);
    //PT.x[id] = fmod(PT.x[id] + (PT.fx[id] * PM.tStep + fT * FRx) / PM.gammaValue + PM.boxX, PM.boxX);
    //PT.y[id] = fmod(PT.y[id] + (PT.fy[id] * PM.tStep + fT * FRy) / PM.gammaValue + PM.boxY, PM.boxY);
    PT.x[id] = fmod(PT.x[id] + (fT * FRx) / PM.gammaValue + PM.boxX, PM.boxX);
    PT.y[id] = fmod(PT.y[id] + (fT * FRy) / PM.gammaValue + PM.boxY, PM.boxY);

}

__device__ real generateNormal(hiprandState* state) {
    return hiprand_normal(&(*state));
}

__device__ int sign(real x) {
    return -(x < 0.f) + (x > 0.f);
}

__device__ int sign01(real x) {
    return (sign(x) + 1) / 2;
}

__global__ void checkUpdate(Particle PT, Parameter PM) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= PM.particleNum)return;
    real x1 = PT.x[id], x0 = PT.x0ToUpdateHybridList[id];
    real y1 = PT.y[id], y0 = PT.y0ToUpdateHybridList[id];
    real dx = sign(x1 - x0) * (x1 - x0);
    real dy = sign(y1 - y0) * (y1 - y0);
    dx = sign01(0.5 * PM.boxX - dx) * dx + sign01(dx - 0.5 * PM.boxX) * (PM.boxX - dx);
    dy = sign01(0.5 * PM.boxY - dy) * dy + sign01(dy - 0.5 * PM.boxY) * (PM.boxY - dy);
    if ((dx * dx + dy * dy) > PM.rOutUpdateList * PM.rOutUpdateList) atomicExch(&updateListFlag, 1);
}

void showProgress(real tNow, real tStart, real tStop, clock_t clockNow, clock_t clockStart) {
    real progress = (tNow - tStart) / (tStop - tStart);
    real tUsed = double(clockNow - clockStart) / CLOCKS_PER_SEC;
    real tUsePrediction = (tStop - tNow) * tUsed / (tNow - tStart);
    printf("First Particle(test Error): %.8f,%.8f\t", pt.x[0], pt.y[0]);
    printf("  Progress:%.4f\%,Prediction:%.1fs\t\r", progress*100, tUsePrediction);
    fflush(stdout);
}

void MemFree() {
    // Free host memory
    delete[] pt.x;
    delete[] pt.y;
    delete[] pt.cellList;
    delete[] pt.cellOffsetsCL;
    delete[] pt.particleAroundId;
    delete[] pt.particleAroundFlagX;
    delete[] pt.particleAroundFlagY;
    delete[] pt.offsetsNL;
    delete[] pt.offsetsAL;
    delete[] pt.NeighborList;
    delete[] pt.NeighborListFlagX;
    delete[] pt.NeighborListFlagY;
    delete[] pt.fx;
    delete[] pt.fy;
    delete[] pt.x0ToUpdateHybridList;
    delete[] pt.y0ToUpdateHybridList;
    delete[] pt.state;
    delete[] pt.aroundNum;
    delete[] pt.kBT;

    // Free device memory
    hipFree(pt.x);
    hipFree(pt.y);
    hipFree(pt.cellX);


    // Free device memory
    hipFree(PT.x);
    hipFree(PT.y);
    hipFree(PT.cellX);
    hipFree(PT.cellY);
    hipFree(PT.cellList);
    hipFree(PT.cellOffsetsCL);
    hipFree(PT.particleAroundId);
    hipFree(PT.particleAroundFlagX);
    hipFree(PT.particleAroundFlagY);
    hipFree(PT.offsetsAL);
    hipFree(PT.offsetsNL);
    hipFree(PT.NeighborList);
    hipFree(PT.NeighborListFlagX);
    hipFree(PT.NeighborListFlagY);
    hipFree(PT.fx);
    hipFree(PT.fy);
    hipFree(PT.x0ToUpdateHybridList);
    hipFree(PT.y0ToUpdateHybridList);
    hipFree(PT.state);
    hipFree(PT.aroundNum);
    hipFree(PT.kBT);
}
